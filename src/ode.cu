#include "hip/hip_runtime.h"
#include <math.h>
#include "ode.h"
#include "model.h"
#include "Random.h"
using namespace std;
/* ------------------------ */
__global__ void evolve(double time, double dt, double tnext, double yy[], hiprandState dev_rand_state[]){
  int tid = blockIdx.x;
  int local_index= tid*pdim;
  double tlocal=time;
  double deltat=dt;
  while(tlocal < tnext){
    stochastic(yy,dev_rand_state,tlocal,deltat,local_index);
    //euler(yy,  tlocal, deltat, local_index); 
    //rnkt2(yy,  tlocal, deltat, local_index);
    rnkt4(yy, tlocal, deltat, local_index);
    tlocal=tlocal+deltat;
  }
}
/*********************************/
__device__ void euler(double yy[], double tt,double deltat, int lindex){
  double k1[pdim];
  eval_rhs(k1,tt,yy,lindex);
  for(int idim=0;idim<pdim;idim++){
    yy[lindex+idim]=yy[lindex+idim]+deltat*k1[idim];
  }
}
/*********************************/
__device__ void rnkt2(double yy[], double tt,double deltat, int lindex){
  double temp[pdim],k1[pdim];
  eval_rhs(k1,tt,yy,lindex);
  for(int idim=0;idim<pdim;idim++){
    temp[idim]=yy[lindex+idim]+k1[idim]*deltat/2.;
  }
  eval_rhs(k1,tt+(deltat/2.),temp,0);
  for(int idim=0;idim<pdim;idim++){
    yy[lindex+idim]=yy[lindex+idim]+deltat*k1[idim];
  }
}
/*********************************/
__device__ void rnkt4(double yy[], double tt,double deltat, int lindex){
  double  temp[pdim],k1[pdim],k2[pdim],k3[pdim],k4[pdim];
  eval_rhs(k1,tt,yy,lindex);
  for(int idim=0;idim<pdim;idim++){
    temp[idim]=yy[lindex+idim]+k1[idim]*deltat/2.;
  }
  eval_rhs(k2,tt+(deltat/2.),temp,0);
  for(int idim=0;idim<pdim;idim++){
    temp[idim]=yy[lindex+idim]+k2[idim]*deltat/2.;
  }
  eval_rhs(k3,tt+(deltat/2.),temp,0);
  for(int idim=0;idim<pdim;idim++){
    temp[idim]=yy[lindex+idim]+k3[idim]*deltat;
  }
  eval_rhs(k4,tt+deltat,temp,0);
  for(int idim=0;idim<pdim;idim++){
    yy[lindex+idim]=yy[lindex+idim]+deltat*(  (k1[idim]/6.) + (k2[idim]/3.) + (k3[idim]/3.) + (k4[idim]/6.) );
  }
}
/*********************************/
