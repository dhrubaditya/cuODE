#include "hip/hip_runtime.h"
#include<math.h>
#include<stdio.h>
#include "CUDA.h"
#include "Random.h"
#include "mycomplex.h"
#include "model.h"
using namespace std;
/* ----------------------------------------*/
/**************************/
  double *yzero;
__device__ double const tau=1.;
__device__ double const one_over_tau=1./tau;
double const iniy_max=10.;
double const iniy_min=-10.;
__device__ inline double kappa(double zz){
  double amp=0.0387;
  double kzero=0.0017;
/*  double zmax=100.;
  double mu=0.1;
  double zabs=abs(zz);
  if (zabs > zmax){
  return mu*zmax*zmax;
  }
  else{
  return mu*zz*zz;
  } */
  return amp*sin(zz)*sin(zz)+kzero;
}
/* ----------------------------------------*/
__global__ void inirand_evolve(unsigned long long seed[]);
/* ----------------------------------------*/
/*__device__ double telegraph(double nu, double tt, int local_index, hiprandState mystate){
  double poisson_mean=nu*tt;
  double pr=Poisson(poisson_mean,&mystate);
  int ppower=(int) fmod(pr,2.);
  double tele_ran = powf(-1,ppower);
  return tele_ran;
}*/
/* ----------------------------------------*/
__device__ void eval_rhs(double rhs[],double tt,double yy[],int lindex){
  double vv=yy[lindex+1];
  double zdot = vv;
/* --The stochastic part of the equation is added outside the usual integrator - */
  double vdot = -one_over_tau*vv;
/* ---------------------------------------------------- */
  rhs[0]=zdot;
  rhs[1]=vdot;
}
/* ----------------------------------------*/
__device__ void stochastic(double yy[],hiprandState global_state[], double tlocal,
         double deltat,int lindex)
{
  double pi = 4.*atan(1.);
  double zz=yy[lindex];
  //double r = fmod(zz,pi);
  double mean=0;
  //double sigma=1.;
  double sigma=kappa(zz);
  int tid=lindex/pdim;
  hiprandState local_state=global_state[tid];
  double uu = Gaussian(mean,sigma,&local_state); 
  global_state[tid] = local_state;
  yy[lindex+1]=yy[lindex+1]+one_over_tau*uu*sqrt(deltat);
}
/*---------------
__global__ void inirand_evolve(unsigned long long seed[], dev_global_state[]){
  int tid = blockIdx.x;
  unsigned long long local_seed = seed[tid];
  hiprandState local_state;
  local_state = dev_global_state[tid];
  hiprand_init(local_seed,tid,0, &local_state);
  dev_global_state[tid] = local_state;
}*/
/* ----------------------------------------*/
void iniconf(double y[],int Nensemble, hiprandState rand_state[]){
  hiprandState *dev_iniran_state;
  double rand[Nensemble],rand2[Nensemble];
  double *dev_rand;
  unsigned long long seed[Nensemble];
  unsigned long long *dev_seed;
  for(int i=0;i<Nensemble;i++){
    seed[i]=37*i+53*i*i;
    rand[i]=0.;
    rand2[i]=0.;
  }
  dev_rand= host2dev(Nensemble,rand);
  dev_seed =  host2dev(Nensemble,seed);
  hipMalloc( (void**)&dev_iniran_state, Nensemble*sizeof(hiprandState) );
  init_random<<<Nensemble,1>>>(dev_seed,dev_iniran_state);
  UniformRandom<<<Nensemble,1>>>(dev_rand, dev_iniran_state);
  dev2host(rand,Nensemble,dev_rand);
  UniformRandom<<<Nensemble,1>>>(dev_rand, dev_iniran_state);
  dev2host(rand2,Nensemble,dev_rand);
  for(int j=0;j<Nensemble;j++){
// Uniformly distributed initial position between iniy_min to iniy_max
      y[0+j*pdim]=iniy_min+rand[j]*(iniy_max-iniy_min);
// and random initial velocity
      y[1+j*pdim]=rand2[j];
      printf("y0,y1,%lf,%lf\n",y[0],y[1]);
  }
  /* copy the state of the random no. generator to host */
  dev2host(rand_state,Nensemble,dev_iniran_state);
  //  inirand_evolve<<<Nensemble,1>>>(dev_seed, dev_rand_state);
}
/* ----------------------------------------*/
__host__ void diag(double tt, double y[], int Nensemble, FILE* tseries, FILE* diagf){
  int ndim=pdim*Nensemble;
  if (tt == 0.) {
     yzero=(double*)malloc(ndim*sizeof(double));
     for (int i=0;i<ndim;i++){
       yzero[i]=y[i];
     }
  }
  //printf("%lf\t%lf\t%lf\t%lf\t%lf\n",tt,y[0],y[1],y[2],y[3]);
  fprintf(tseries,"%lf\t",tt);
  for (int i=0;i<ndim-1;i++){
    fprintf(tseries,"%lf\t",y[i]);
  }
  fprintf(tseries,"%lf\n",y[ndim-1]);
  double meanz=0.;
  double meanv=0.;
  double dzrms=0;
  for(int i=0; i<Nensemble; i++){
    int lindex=pdim*i;
    double zz=y[lindex];
    double dz=y[lindex]-yzero[lindex];
    double vv=y[lindex+1];
    meanz= zz+meanz ;
    dzrms= dz*dz+dzrms ;
    meanv= vv+meanv ;
  }
  meanz=meanz/Nensemble;
  meanv=meanv/Nensemble;
  dzrms=sqrt(dzrms)/Nensemble;
  printf("%lf\t%lf\t%lf\t%lf\n",tt,dzrms,meanz,meanv);
  fprintf(diagf,"%lf\t%lf\t%lf\t%lf\n",tt,dzrms,meanz,meanv);
}
/* ----------------------------------------*/
