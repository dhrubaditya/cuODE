#include <iostream>
#include<fstream>
#include "CUDA.h"
#include "Random.h"
#include "model.h"
using namespace std;
/**************************/
__device__ double const omega2=1.;
/* ----------------------------------------*/
__device__ void stochastic(double yy[],hiprandState dev_rand_state[], double tlocal,
         double deltat,int lindex){}
/* ----------------------------------------*/
__device__ void eval_rhs(double rhs[],double tt,double yy[],int lindex){
  /* we solve:
        (d/dt)x = v;  (d/dt)v = -\omega^2 x
 */
  double xx=yy[lindex];
  double vv=yy[lindex+1];
  rhs[0]=vv;
  rhs[1]=-omega2*xx;
}
/* ----------------------------------------*/
__host__ void iniconf(double y[],int Nensemble, hiprandState rand_state[]){
  double rand[Nensemble];
  double *dev_rand;
  hiprandState *dev_iniran_state;
  unsigned long long seed[Nensemble];
  unsigned long long *dev_seed;
  for(int i=0;i<Nensemble;i++){
    seed[i]=37*i+53*i*i;
    rand[i]=0.;
  }
  dev_rand= host2dev(Nensemble,rand);
  dev_seed =  host2dev(Nensemble,seed);
  hipMalloc( (void**)&dev_iniran_state, Nensemble*sizeof(hiprandState) );
  init_random<<<Nensemble,1>>>(dev_seed,dev_iniran_state);
  UniformRandom<<<Nensemble,1>>>(dev_rand, dev_iniran_state);
  dev2host(rand,Nensemble,dev_rand);
  for(int j=0;j<Nensemble;j++){
    y[0+j*pdim]=rand[j];
  }
  UniformRandom<<<Nensemble,1>>>(dev_rand, dev_iniran_state);
  dev2host(rand,Nensemble,dev_rand);
  for(int j=0;j<Nensemble;j++){
    y[1+j*pdim]=rand[j];
  }
  dev2host(rand_state,Nensemble,dev_iniran_state);
}
/* ----------------------------------------*/
__host__ void diag(double tt, double y[], int Nensemble, FILE* tseries, FILE* diagf){
  int ndim=pdim*Nensemble;
  printf("%lf\t%lf\t%lf\t%lf\t%lf\n",tt,y[0],y[1],y[2],y[3]);
  fprintf(tseries,"%lf\t",tt);
  for (int i=0;i<ndim-1;i++){
    fprintf(tseries,"%lf\t",y[i]);
  }
  fprintf(tseries,"%lf\n",y[ndim-1]);
  fprintf(diagf,"%lf\t",tt);
  for(int i=0; i<Nensemble; i++){
    double xx=y[i*pdim+0];
    double vv=y[i*pdim+1];
    double energy=omega2*xx*xx+vv*vv;
    fprintf(diagf,"%lf\t",energy);
  }
  fprintf(diagf,"\n");
}
/* ----------------------------------------*/




