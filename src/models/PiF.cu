#include "hip/hip_runtime.h"
#include<math.h>
#include<stdio.h>
#include "CUDA.h"
#include "Random.h"
#include "mycomplex.h"
#include "model.h"
using namespace std;
/* ----------------------------------------*/
/**************************/
  double *yzero;
__device__ double const tau=1.;
__device__ double const one_over_tau=1./tau;
double const iniy_max=10.;
double const iniy_min=-10.;
__device__ inline double kappa(double zz){
  double amp=1.;
/*  double zmax=100.;
  double mu=0.1;
  double zabs=abs(zz);
  if (zabs > zmax){
  return mu*zmax*zmax;
  }
  else{
  return mu*zz*zz;
  } */
  return amp;
}
/* ----------------------------------------*/
__global__ void inirand_evolve(unsigned long long seed[]);
/* ----------------------------------------*/
/*__device__ double telegraph(double nu, double tt, int local_index, hiprandState mystate){
  double poisson_mean=nu*tt;
  double pr=Poisson(poisson_mean,&mystate);
  int ppower=(int) fmod(pr,2.);
  double tele_ran = powf(-1,ppower);
  return tele_ran;
}*/
/* ----------------------------------------*/
__device__ void eval_rhs(double rhs[],double tt,double yy[],int lindex){
  double v1=yy[lindex+3];
  double v2=yy[lindex+4];
  double v3=yy[lindex+5];
  double zdot1 = v1;
  double zdot2 = v2;
  double zdot3 = v3;
/* --The stochastic part of the equation is added outside the usual integrator - */
  double vdot1 = -one_over_tau*v1;
  double vdot2 = -one_over_tau*v2;
  double vdot3 = -one_over_tau*v3;
/* ---------------------------------------------------- */
  rhs[0]=zdot1;
  rhs[1]=zdot2;
  rhs[2]=zdot3;
  rhs[3]=vdot1;
  rhs[4]=vdot2;
  rhs[5]=vdot3;
}
/* ----------------------------------------*/
__device__ void stochastic(double yy[],hiprandState global_state[], double tlocal,
         double deltat,int lindex)
{
  double pi = 4.*atan(1.);
  //double zz=yy[lindex];
  //double r = fmod(zz,pi);
  double mean=0;
  double sigma=1.;
  //double sigma=kappa(zz);
  int tid=lindex/pdim;
  hiprandState local_state=global_state[tid];
  double uu1 = Gaussian(mean,sigma,&local_state); 
  double uu2 = Gaussian(mean,sigma,&local_state); 
  double uu3 = Gaussian(mean,sigma,&local_state); 
  global_state[tid] = local_state;
  yy[lindex+3]=yy[lindex+3]+one_over_tau*uu1*sqrt(deltat);
  yy[lindex+4]=yy[lindex+4]+one_over_tau*uu2*sqrt(deltat);
  yy[lindex+5]=yy[lindex+5]+one_over_tau*uu3*sqrt(deltat);
}
/*---------------
__global__ void inirand_evolve(unsigned long long seed[], dev_global_state[]){
  int tid = blockIdx.x;
  unsigned long long local_seed = seed[tid];
  hiprandState local_state;
  local_state = dev_global_state[tid];
  hiprand_init(local_seed,tid,0, &local_state);
  dev_global_state[tid] = local_state;
}*/
/* ----------------------------------------*/
void iniconf(double y[],int Nensemble, hiprandState rand_state[]){
  hiprandState *dev_iniran_state;
  double rand[Nensemble],rand2[Nensemble];
  double *dev_rand;
  unsigned long long seed[Nensemble];
  unsigned long long *dev_seed;
  for(int i=0;i<Nensemble;i++){
    seed[i]=37*i+53*i*i;
    rand[i]=0.;
    rand2[i]=0.;
  }
  dev_rand= host2dev(Nensemble,rand);
  dev_seed =  host2dev(Nensemble,seed);
  hipMalloc( (void**)&dev_iniran_state, Nensemble*sizeof(hiprandState) );
  init_random<<<Nensemble,1>>>(dev_seed,dev_iniran_state);
  UniformRandom<<<Nensemble,1>>>(dev_rand, dev_iniran_state);
  dev2host(rand,Nensemble,dev_rand);
  UniformRandom<<<Nensemble,1>>>(dev_rand, dev_iniran_state);
  dev2host(rand2,Nensemble,dev_rand);
  for(int j=0;j<Nensemble;j++){
// Uniformly distributed initial position between iniy_min to iniy_max
      y[0+j*pdim]=iniy_min+rand[j]*(iniy_max-iniy_min);
// and random initial velocity
      y[1+j*pdim]=rand2[j];
      printf("y0,y1,%lf,%lf\n",y[0],y[1]);
  }
  /* copy the state of the random no. generator to host */
  dev2host(rand_state,Nensemble,dev_iniran_state);
  //  inirand_evolve<<<Nensemble,1>>>(dev_seed, dev_rand_state);
}
/* ----------------------------------------*/
__host__ void diag(double tt, double y[], int Nensemble, FILE* tseries, FILE* diagf){
  int ndim=pdim*Nensemble;
  if (tt == 0.) {
     yzero=(double*)malloc(ndim*sizeof(double));
     for (int i=0;i<ndim;i++){
       yzero[i]=y[i];
     }
  }
  //printf("%lf\t%lf\t%lf\t%lf\t%lf\n",tt,y[0],y[1],y[2],y[3]);
  fprintf(tseries,"%lf\t",tt);
  for (int i=0;i<ndim-1;i++){
    fprintf(tseries,"%lf\t",y[i]);
  }
  fprintf(tseries,"%lf\n",y[ndim-1]);
  double meanz=0.;
  double vsqrm=0.;
  double dzrms=0;
  for(int i=0; i<Nensemble; i++){
    int lindex=pdim*i;
    double zz1=y[lindex+0];
    double zz2=y[lindex+1];
    double zz3=y[lindex+2];
    double vv1=y[lindex+3];
    double vv2=y[lindex+4];
    double vv3=y[lindex+5];
    double dz=zz1-yzero[lindex];
    meanz= zz1+meanz ;
    dzrms= dz*dz+dzrms ;
    vsqrm= vv1*vv1+vv2*vv2+vv3*vv3+vsqrm;
  }
  double zz=y[0];
  meanz=meanz/Nensemble;
  vsqrm=vsqrm/Nensemble;
  dzrms=sqrt(dzrms)/Nensemble;
  printf("%lf\t%lf\t%lf\t%lf\n",tt,dzrms,meanz,vsqrm);
  //printf("%lf\t%lf\t%lf\n",tt,u1,u2);
  fprintf(diagf,"%lf\t%lf\t%lf\t%lf\n",tt,zz,dzrms,meanz,vsqrm);
}
/* ----------------------------------------*/
