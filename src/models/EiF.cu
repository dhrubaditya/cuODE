#include <iostream>
#include<fstream>
#include "CUDA.h"
#include "Random.h"
#include "modules/quaternion.h"
#include "modules/matrix.h"
#include "modules/RigidBody.h"
#include "model.h"
using namespace std;
/**************************/
__device__ double const omega2=1.;
__device__ unsigned int const ndim=pdim*Nensemble;
vec3 inipos(int posini);
__device__ vec3 inivel(int velini);
__device__ quaternion iniq(int qini);
vec3 iniell(int ellini, quaternion qq_zero);
vec3 fluid_velocity(vec3 );
__device__ void U_Omega_GradU(vec3& UU, vec3& OO, double GradU[3][3], vec3 xx);
__device__ vec3 DragForce(vec3 UU,RigidBody *rb);
__device__ vec3 JefferyTorque(vec3 OO,double Sij[3][3],RigidBody *rb);
__host__ void wparam_sim(double time);
/* ----------------------------------------*/
__device__ void stochastic(double yy[],hiprandState dev_rand_state[], double tlocal,
         double deltat,int lindex){}
/* ----------------------------------------*/
__device__ void eval_rhs(double rhs[],double tt,double yy[],int lindex){
  /* we solve:
        Equations of a rigid body advected by known flow
 */
  double xx=yy[lindex];
  double vv=yy[lindex+1];


  RigidBody rb;
  double GradU[3][3],Sij[3][3];
  vec3 UU,OO,vv,Force,Torque;
  quaternion omegaq,dq_dt;
  //
  for(int ibody=0;ibody<Nensemble;ibody++){
    int irb=pdim*ibody;
    array2rb(&rb, &y[irb]);
    U_Omega_GradU(UU, OO, GradU, rb.xx);
    for(int i=0;i<3;i++){
      for(int j=0;j<3;j++){
        Sij[i][j]= 0.5*(GradU[i][j]+GradU[j][i]);
      }
    }
    /* vec3 UXO = cross(UU,OO);
    cout<<"U, O, UXO"<<"\n";g
    PVec3(UU);
    PVec3g(OO);
    PVec3(UXO);
    cout<<"-----------------------"<<"\n"; */
//----- Calculate velocity -------
    vv = rb.pp*(1./Mass);
//---Calculate Force---------
    Force=DragForce(UU,&rb);
    //PVec3(Force);
// -- Now for zero torque -----------
    Torque=JefferyTorque(OO,Sij,&rb);
/* ---- Calculate the evolution eqn for the quaternion 
   dq_dt = (1/2) omega * q  */
    omegaq = quaternion(0.,rb.omega);
    dq_dt = omegaq * rb.qq *(1./2.);

// ----------------------------------
    rhs[irb+0]=rb.pp.x/Mass;
    rhs[irb+1]=rb.pp.y/Mass;
    rhs[irb+2]=rb.pp.z/Mass;
    //
    rhs[irb+3]=Force.x;
    rhs[irb+4]=Force.y;
    rhs[irb+5]=Force.z;
    //
    rhs[irb+6]=dq_dt.w;
    rhs[irb+7]=dq_dt.u.x;
    rhs[irb+8]=dq_dt.u.y;
    rhs[irb+9]=dq_dt.u.z;
    //
    rhs[irb+10]=Torque.x;
    rhs[irb+11]=Torque.y;
    rhs[irb+12]=Torque.z;
    }







  rhs[0]=vv;
  rhs[1]=-omega2*xx;
}
/* ----------------------------------------*/
__host__ void iniconf(double y[],int Nensemble, hiprandState rand_state[]){
  double rand[Nensemble];
  double *dev_rand;
  hiprandState *dev_iniran_state;
  unsigned long long seed[Nensemble];
  unsigned long long *dev_seed;
  for(int i=0;i<Nensemble;i++){
    seed[i]=37*i+53*i*i;
    rand[i]=0.;
  }
  dev_rand= host2dev(Nensemble,rand);
  dev_seed =  host2dev(Nensemble,seed);
  hipMalloc( (void**)&dev_iniran_state, Nensemble*sizeof(hiprandState) );
  init_random<<<Nensemble,1>>>(dev_seed,dev_iniran_state);
  UniformRandom<<<Nensemble,1>>>(dev_rand, dev_iniran_state);
  dev2host(rand,Nensemble,dev_rand);
  for(int j=0;j<Nensemble;j++){
    y[0+j*pdim]=rand[j];
  }
  UniformRandom<<<Nensemble,1>>>(dev_rand, dev_iniran_state);
  dev2host(rand,Nensemble,dev_rand);
  for(int j=0;j<Nensemble;j++){
    y[1+j*pdim]=rand[j];
  }
  dev2host(rand_state,Nensemble,dev_iniran_state);
}
/* ----------------------------------------*/
__host__ void diag(double tt, double y[], int Nensemble, FILE* tseries, FILE* diagf){
  int ndim=pdim*Nensemble;
  printf("%lf\t%lf\t%lf\t%lf\t%lf\n",tt,y[0],y[1],y[2],y[3]);
  fprintf(tseries,"%lf\t",tt);
  for (int i=0;i<ndim-1;i++){
    fprintf(tseries,"%lf\t",y[i]);
  }
  fprintf(tseries,"%lf\n",y[ndim-1]);
  fprintf(diagf,"%lf\t",tt);
  for(int i=0; i<Nensemble; i++){
    double xx=y[i*pdim+0];
    double vv=y[i*pdim+1];
    double energy=omega2*xx*xx+vv*vv;
    fprintf(diagf,"%lf\t",energy);
  }
  fprintf(diagf,"\n");
}
/* ----------------------------------------*/




