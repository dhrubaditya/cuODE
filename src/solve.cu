#include <iostream>
#include<fstream>
#include<stdlib.h>
#include<time.h>
#include<math.h>
#include "ode.h"
#include "input.h"
#include "model.h"
#include "CUDA.h"
#include "Random.h"
#include "SpecialFunctions.h"
using namespace std;
/**************************/
/* ----------------------------------------*/
int main(){
  //--------------------------------------//
  unsigned int const ndim=Nensemble*pdim;
  hiprandState rand_state[Nensemble];
  hiprandState *dev_rand_state;
  double y[ndim];
/* ---- to debug the noise --------*/
  double noise[Nensemble];
  double *dev_noise;
  for(int i=0;i<Nensemble;i++){
    noise[i]=0.;
  }
 dev_noise=host2dev(Nensemble,noise); 
/*----------------------------------*/
  double time=0.;
  double tnext;
  /* device variables */
  double *dev_y;
  /* end device variables */
  iniconf(y, Nensemble, rand_state);
/*---------------------------------*/
  dev_y=host2dev(ndim,y);
  dev_rand_state=host2dev(Nensemble,rand_state);
  double dtnext=TMAX/Ndiag;
  tnext=dtnext;
  /*----------------------------------------------------------------*/
  FILE *tseries=fopen("tseries.out","w");
  FILE *diagf=fopen("diag.dat","w");
  while (time < TMAX){
    diag(time,y,Nensemble,tseries,diagf);
    evolve<<<Nensemble,1>>>(time,dt,tnext,dev_y,dev_rand_state);
    dev2host(y,ndim,dev_y); 
    dev2host(rand_state,Nensemble,dev_rand_state);
    dev2host(noise,Nensemble,dev_noise);
    time=time+dtnext;
    tnext=time+dtnext;
    printf("Next diagnostic at %f\n",tnext);
  }
  fclose(tseries);
  fclose(diagf);
  FILE *fin_state=fopen("yfin.out","w");
  fprintf(fin_state,"%lf\n",time);
  for (int i=0;i<ndim;i++){
    fprintf(fin_state,"%lf\n",y[i]);
  }
  fclose(fin_state);
//----------------------------
}
/* ----------------------------------------*/
