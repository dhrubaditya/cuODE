#include "CUDA.h"
/* ------------------------ */
double *allocdev_double(int len){
  double *dev_array;
  hipMalloc( (void**)&dev_array, len*sizeof(double) );
  return dev_array;
}
int *allocdev_int(int len){
  int *dev_array;
  hipMalloc( (void**)&dev_array, len*sizeof(int) );
  return dev_array;
}
hiprandState *host2dev(int len, hiprandState host_array[]){
  hiprandState *dev_array;
  hipMalloc( (void**)&dev_array, len*sizeof(hiprandState) );
  hipMemcpy(dev_array, host_array, len*sizeof(hiprandState), hipMemcpyHostToDevice);
  return dev_array;
}
double *host2dev(int len, double host_array[]){
  double *dev_array;
  hipMalloc( (void**)&dev_array, len*sizeof(double) );
  hipMemcpy(dev_array, host_array, len*sizeof(double), hipMemcpyHostToDevice);
  return dev_array;
}
int *host2dev(int len, int host_array[]){
  int *dev_array;
  hipMalloc( (void**)&dev_array, len*sizeof(int) );
  hipMemcpy(dev_array, host_array, len*sizeof(int), hipMemcpyHostToDevice);
  return dev_array;
}
unsigned long long  *host2dev(int len, unsigned long long host_array[]){
  unsigned long long *dev_array;
  hipMalloc( (void**)&dev_array, len*sizeof(unsigned long long) );
  hipMemcpy(dev_array, host_array, len*sizeof(unsigned long long), hipMemcpyHostToDevice);
  return dev_array;
}
void  h2d(double *dev_array, int len, double host_array[]){
  hipMemcpy(dev_array, host_array, len*sizeof(double), hipMemcpyHostToDevice);
}
void  h2d(int *dev_array, int len, int host_array[]){
  hipMemcpy(dev_array, host_array, len*sizeof(int), hipMemcpyHostToDevice);
}
void dev2host(hiprandState host_array[], int len, hiprandState *dev_array){
  hipMemcpy(host_array, dev_array, len*sizeof(hiprandState),hipMemcpyDeviceToHost);
}
void dev2host(double host_array[], int len, double *dev_array){
  hipMemcpy(host_array, dev_array, len*sizeof(double),hipMemcpyDeviceToHost);
}
void dev2host(int host_array[], int len, int *dev_array){
  hipMemcpy(host_array, dev_array, len*sizeof(int),hipMemcpyDeviceToHost);
}
void dev2host(unsigned long long host_array[], int len, unsigned long long *dev_array){
  hipMemcpy(host_array, dev_array, len*sizeof(unsigned long long),hipMemcpyDeviceToHost);
}
/*---------------------------------------*/
