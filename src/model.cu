#include "hip/hip_runtime.h"
#include<math.h>
#include "CUDA.h"
#include "Random.h"
#include "mycomplex.h"
#include "model.h"
using namespace std;
/* ----------------------------------------*/
/**************************/
__device__ double const kk=1.;
__device__ double ksqr=kk*kk;
__device__ double const alpha0=1.;
__device__ double const tauc=0.1;
__device__ double nu = 1./tauc;
__device__ double const Shear=-4.;
__device__ double const eta=1.;
/* ------------------------------ */
double Biamp = 1.e-5;
struct Bfield{
  /* State variables */
  complex x; // x component of magnetic field
  complex y; // y component of magnetic field
};
/* ----------------------------------------*/
__global__ void inirand_evolve(unsigned long long seed[]);
__host__ __device__ void array2b(Bfield *B,  double y[]);
__host__ __device__ void b2array(double y[], Bfield *B);
/* ----------------------------------------*/
__device__ double telegraph(double nu, double tt, int local_index, hiprandState mystate){
  double poisson_mean=nu*tt;
  double pr=Poisson(poisson_mean,&mystate);
  int ppower=(int) fmod(pr,2.);
  double tele_ran = powf(-1,ppower);
  return tele_ran;
}
/* ----------------------------------------*/
__device__ void eval_rhs(double rhs[],double tt,double yy[],int lindex){
  Bfield B,dtB;
  complex I=complex(0.,1.);
//
  array2b(&B, &yy[lindex]);
/* --The stochastic part of the equation is added outside the usual integrator - */
  dtB.x= complex(0.,0.) - B.x*(eta*ksqr);
  dtB.y= B.x*Shear -B.y*(eta*ksqr);
// ----------------------------------
  b2array(&rhs[0],&dtB);
//
}
/* ----------------------------------------*/
__device__ void stochastic(double yy[],hiprandState rstate, double tlocal,
         double deltat,int lindex)
{
  Bfield B,dtB;
  complex I=complex(0.,1.);
  array2b(&B, &yy[lindex]);
  double alpha=alpha0*telegraph(nu,tlocal,lindex,rstate);
  B.x= B.x+I*B.y*(kk*alpha)*sqrt(deltat);
  /* B.x=complex(alpha,0.);
  B.y=complex(alpha,0.); */
  b2array(&yy[lindex],&B);
}
/*---------------
__global__ void inirand_evolve(unsigned long long seed[], dev_global_state[]){
  int tid = blockIdx.x;
  unsigned long long local_seed = seed[tid];
  hiprandState local_state;
  local_state = dev_global_state[tid];
  hiprand_init(local_seed,tid,0, &local_state);
  dev_global_state[tid] = local_state;
}*/
/* ----------------------------------------*/
void iniconf(double y[],int Nensemble, hiprandState rand_state[]){
  hiprandState *dev_iniran_state;
  double rand[Nensemble];
  double *dev_rand;
  unsigned long long seed[Nensemble];
  unsigned long long *dev_seed;
  for(int i=0;i<Nensemble;i++){
    seed[i]=37*i+53*i*i;
    rand[i]=0.;
  }
  dev_rand= host2dev(Nensemble,rand);
  dev_seed =  host2dev(Nensemble,seed);
  hipMalloc( (void**)&dev_iniran_state, Nensemble*sizeof(hiprandState) );
  init_random<<<Nensemble,1>>>(dev_seed,dev_iniran_state);
  for(int i=0;i<pdim;i++){
    UniformRandom<<<Nensemble,1>>>(dev_rand, dev_iniran_state);
    dev2host(rand,Nensemble,dev_rand);
    for(int j=0;j<Nensemble;j++){
      y[i+j*pdim]=rand[j]*Biamp;
    }
  }
  /* copy the state of the random no. generator to host */
  dev2host(rand_state,Nensemble,dev_iniran_state);
  //  inirand_evolve<<<Nensemble,1>>>(dev_seed, dev_rand_state);
}
/* ----------------------------------------*/
__host__ void diag(double tt, double y[], int Nensemble, FILE* tseries, FILE* diagf){
  Bfield B;
  int ndim=pdim*Nensemble;
  //printf("%lf\t%lf\t%lf\t%lf\t%lf\n",tt,y[0],y[1],y[2],y[3]);
  fprintf(tseries,"%lf\t",tt);
  for (int i=0;i<ndim-1;i++){
    fprintf(tseries,"%lf\t",y[i]);
  }
  fprintf(tseries,"%lf\n",y[ndim-1]);
  fprintf(diagf,"%lf\t",tt);
  double meanBxreal=0.;
  for(int i=0; i<Nensemble; i++){
    int lindex=pdim*(Nensemble-1);
    array2b(&B, &y[lindex]);
    meanBxreal= B.x.real+meanBxreal ;
    printf("%lf\t%lf\n",tt,meanBxreal);
    fprintf(diagf,"%lf\t",meanBxreal);
  }
  fprintf(diagf,"\n");
}
/* ----------------------------------------*/
__host__ __device__ void array2b(Bfield *B,  double y[]){
  /* real and imaginary part of Bx */
  B->x=complex(y[0],y[1]);
  /* real and imaginary part of By */
  B->y=complex(y[2],y[3]);
}
/* ----------------------------------------*/
__host__ __device__ void b2array(double y[], Bfield *B){
  y[0]=B->x.real;
  y[1]=B->x.imag;
  y[2]=B->y.real;
  y[3]=B->y.imag;
}

