#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "SpecialFunctions.h"
#include "Random.h"
/*------------ Uses CUDA random number generator -------- */
__global__ void init_random(unsigned long long *seed, hiprandState  *global_state){
  int tid = blockIdx.x;
  unsigned long long local_seed = seed[tid];
  hiprandState local_state;
  local_state = global_state[tid];
  hiprand_init(local_seed,tid,0, &local_state);
  global_state[tid] = local_state;
}
/*--------------------------------------*/
__global__ void random(double *x, hiprandState *global_state){
  int tid =  blockIdx.x;
  hiprandState local_state;
  local_state = global_state[tid];
  x[tid] = (double) hiprand(&local_state);
  global_state[tid] = local_state;
}
/*--------------------------------------*/
__global__ void UniformRandom(double *x, hiprandState *global_state){
  int tid =  blockIdx.x;
  hiprandState local_state;
  local_state = global_state[tid];
  x[tid] = (double) hiprand_uniform(&local_state);
  global_state[tid] = local_state;
}
/*--------------------------------------*/
__device__ double Gaussian(double mean, double sigma, hiprandState *mystate){
  double xx= (double) hiprand_normal(mystate);
  double yy=mean+sigma*xx;
  return yy;
}
/*--------------------------------------*/
__device__ double Poisson(double xmean, hiprandState *mystate){
  double reject_factor=0.9,reject;
  double pi;
  pi = 4.*atan(1.);
  double x,xcomp;
  if (xmean < 12.){
    x=-1.;
    double exp_nxm=exp(-xmean);
    double uni_var_product=1.;
    while(uni_var_product > exp_nxm){
      x=x+1.;
      double rand = (double) hiprand_uniform(mystate);
      uni_var_product=uni_var_product*rand;
    }
  }else{
   double sq = sqrt(2.0*xmean);
   double log_xmean = log(xmean);
   double GG = xmean*log_xmean - LnGamma(xmean+1.0);
   do {
     do {
       double rand = (double) hiprand_uniform(mystate);
       xcomp = tan(pi*rand);
       x = sqrt(2.*xmean)*xcomp + xmean;
     } while (x < 0.0);
     x = floor(x);
     reject = reject_factor*(1.0 + xcomp*xcomp)*exp(x*log_xmean - LnGamma(x+1.0) - GG);
   } while (hiprand_uniform(mystate) > reject);
  }
  return x;
}
