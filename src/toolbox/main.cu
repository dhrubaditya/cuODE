#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include "CUDA.h"
#include "SpecialFunctions.h"
#include "Random.h"
#define pen 2
/* ------------------------ */
__global__ void evolve(double *a,double *pr,
                       double *tt,double *deltat, 
                       double *tnxt,double *nuu,
                       double *a0, 
                       hiprandState *global_state){
    int tid = blockIdx.x;
    hiprandState mystate=global_state[tid];
    if (tid < pen) {
      while(*tt < *tnxt){
        double poisson_mean=(*nuu)*(*tt);
        pr[tid]=Poisson(poisson_mean,&mystate);
        int ppower=(int) fmod(pr[tid],2.);
        a[tid] = (*a0)*powf(-1,ppower);
        *tt=*tt+*deltat;
      }
    }
    global_state[tid] = mystate;
}
/* ------------------------ */
int main(void){
  double alpha[pen],prand[pen];
  double *dev_alpha,*dev_prand;
  unsigned long long seed[pen];
  unsigned long long *dev_seed;
  hiprandState *dev_global_state;
  double dt=1e-4;
  double *dev_dt;
  double TMAX=4.;
  double tnext;
  double *dev_tnext;
  double t=0.;
  double *dev_t;
  double alpha0=1.;
  double *dev_alpha0;
  double tauc=0.01;
  double nu=1./tauc;
  double *dev_nu;
  int Ndiag=10;
/* ------- set host values ----------- */
  seed[0]=37;
  seed[1]=53;
  dev_seed =  host2dev(pen,seed);
  hipMalloc( (void**)&dev_global_state, pen*sizeof(hiprandState) );
  init_random<<<pen,1>>>(dev_seed,dev_global_state);
  alpha[0]=1.;
  alpha[1]=-1.;
  dev_alpha = host2dev(pen,alpha);
  prand[0]=0.;
  prand[1]=1.;
  dev_prand = host2dev(pen,prand);
/*---------------------------------*/
  dev_t = host2dev(1,&t);
  dev_dt = host2dev(1,&dt);
  double dtnext=TMAX/Ndiag;
  tnext=dtnext;
  dev_tnext = host2dev(1,&tnext);
  dev_nu = host2dev(1,&nu);
  dev_alpha0 = host2dev(1,&alpha0);
/*----------------------------------------------------------------*/
  while (t < TMAX){
    evolve<<<pen,1>>>(dev_alpha,dev_prand,dev_t,dev_dt,dev_tnext,dev_nu,dev_alpha0,dev_global_state);
   dev2host(alpha,pen,dev_alpha); 
   dev2host(prand,pen,dev_prand); 
   t=t+dtnext;
   h2d(dev_t,1,&t);
   tnext=t+dtnext;
   h2d(dev_tnext,1,&tnext);
   printf("%lf\t%lf\t%lf\n",t,prand[0],alpha[0]);
   printf("Next diagnostic at %f\n",tnext);
 }
  
}
