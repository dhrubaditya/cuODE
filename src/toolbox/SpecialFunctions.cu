#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "SpecialFunctions.h"
/*---------------------------------------*/
__device__ double LnGamma(double zp1){
  /* Calculates the logarithm of Gamma function using Lancsoz algorithm, 
     following the code given in Numerical Recepies (fortran) page 206-207 
     \Gamma(zz+1) = (zz+gamma+1/2)^{z+1/2}e^{-(z+gamma+1/2)}\sqrt{2\pi}[c_0+\sum_{j=1,6} \frac{c_j}{z+j}]
     ln\Gamma(zz+1) = (zz+1/2)ln(zz+gamma+1/2)+[-(z+gamma+1/2)] + (0.5)\ln(2\pi)+\ln[c_0+\sum_{j=1,6} \frac{c_j}{z+j}]]
     ln\Gamma(zp1) = (zp1-1/2)ln(zp1-1/2+gamma)+[-(zp1-1/2+gamma)] + (0.5)\ln(2\pi)+\ln[c_0+\sum_{j=1,6} \frac{c_j}{zp1+j-1}]]
*/
  double LG;
  int const N=6;
  double zz=zp1-1;
  if (zz >= 0 ){
    double pi;
    pi = 4.*atan(1.);
    double sqrt2pi=sqrt(2.*pi);
    double cof[N+1];
    cof[0]=1.000000000190015;
    cof[1]=76.18009171947146;
    cof[2]=-86.50532032941677;
    cof[3]=24.01409824083091;
    cof[4]=-1.231739572450155;
    cof[5]=0.1208650973866179e-2;
    cof[6]=-0.5395239384953e-5;
    double gamma=5.;
    double sum=0.;
    for (int j=1; j<=N; j++){
      sum = sum+cof[j]/(zz+(double)(j));
    }
    sum = sum + cof[0];
    double lnsum=log(sqrt2pi*sum);
    LG= (zz+0.5)*log(zz+gamma+0.5)-(zz+gamma+0.5) +lnsum ;
  }else{
    printf("Routine calculates Log Gamma function only for positive argument\n");
    LG=0.;
  }
    return LG;
}
/*---------------------------------------*/

